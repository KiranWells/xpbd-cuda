#include "hip/hip_runtime.h"
/******************************************************************************

  This source code accompanies the Journal of Graphics Tools paper:

  "Fast Ray-Axis Aligned Bounding Box Overlap Tests With Pluecker Coordinates"
 by Jeffrey Mahovsky and Brian Wyvill Department of Computer Science, University
 of Calgary

  This source code is public domain, but please mention us if you use it.

 ******************************************************************************/

#include "JGT-float/smits_mul.h"

namespace jgt_float {

bool smits_mul(ray *r, aabox *b, float *t) {
  float tnear = -1e6;
  float tfar = 1e6;

  {
    // multiply by the inverse instead of dividing
    float t1 = (b->x0 - r->x) * r->ii;
    float t2 = (b->x1 - r->x) * r->ii;

    if (t1 > t2) {
      float temp = t1;
      t1 = t2;
      t2 = temp;
    }
    if (t1 > tnear)
      tnear = t1;
    if (t2 < tfar)
      tfar = t2;

    if (tnear > tfar)
      return false;
    if (tfar < 0.0)
      return false;
  }
  {
    float t1 = (b->y0 - r->y) * r->ij;
    float t2 = (b->y1 - r->y) * r->ij;

    if (t1 > t2) {
      float temp = t1;
      t1 = t2;
      t2 = temp;
    }
    if (t1 > tnear)
      tnear = t1;
    if (t2 < tfar)
      tfar = t2;

    if (tnear > tfar)
      return false;
    if (tfar < 0.0)
      return false;
  }
  {
    float t1 = (b->z0 - r->z) * r->ik;
    float t2 = (b->z1 - r->z) * r->ik;

    if (t1 > t2) {
      float temp = t1;
      t1 = t2;
      t2 = temp;
    }
    if (t1 > tnear)
      tnear = t1;
    if (t2 < tfar)
      tfar = t2;

    if (tnear > tfar)
      return false;
    if (tfar < 0.0)
      return false;
  }

  *t = tnear;
  return true;
}

} // namespace jgt_float
