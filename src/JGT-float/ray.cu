/******************************************************************************

  This source code accompanies the Journal of Graphics Tools paper:

  "Fast Ray-Axis Aligned Bounding Box Overlap Tests With Pluecker Coordinates"
 by Jeffrey Mahovsky and Brian Wyvill Department of Computer Science, University
 of Calgary

  This source code is public domain, but please mention us if you use it.

 ******************************************************************************/

#include "JGT-float/ray.h"

namespace jgt_float {

void make_ray(float x, float y, float z, float i, float j, float k, ray *r) {
  r->x = x;
  r->y = y;
  r->z = z;
  r->i = i;
  r->j = j;
  r->k = k;
  r->ii = 1.0f / i;
  r->ij = 1.0f / j;
  r->ik = 1.0f / k;
  r->R0 = x * j - i * y;
  r->R1 = x * k - i * z;
  r->R3 = y * k - j * z;

  if (i < 0) {
    if (j < 0) {
      if (k < 0)
        r->classification = MMM;
      else
        r->classification = MMP;
    } else {
      if (k < 0)
        r->classification = MPM;
      else
        r->classification = MPP;
    }
  } else {
    if (j < 0) {
      if (k < 0)
        r->classification = PMM;
      else
        r->classification = PMP;
    } else {
      if (k < 0)
        r->classification = PPM;
      else
        r->classification = PPP;
    }
  }
}

} // namespace jgt_float
