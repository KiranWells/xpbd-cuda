#include "util.h"

void *alloc_device_bytes(size_t bytes) {
#ifdef USE_CUDA
  void *device_ptr;
  CUDA_CHECK(hipMalloc(&device_ptr, bytes));
  return device_ptr;
#else
  return malloc(bytes);
#endif
}

void memcpy_device(void *__restrict dest, void *__restrict src, size_t bytes) {
#ifdef USE_CUDA
  CUDA_CHECK(hipMemcpy(dest, src, bytes, hipMemcpyDeviceToDevice));
#else
  memcpy(dest, src, bytes);
#endif
}

void memcpy_host_device(void *__restrict dest, void *__restrict src,
                        size_t bytes) {
#ifdef USE_CUDA
  CUDA_CHECK(hipMemcpy(dest, src, bytes, hipMemcpyHostToDevice));
#else
  memcpy(dest, src, bytes);
#endif
}
