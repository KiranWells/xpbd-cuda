#include "hip/hip_runtime.h"
#include "apbd/Model.h"
#include "util.h"

#include <iostream>
#include <stdexcept>

namespace apbd {

Model::Model()
    : h(1. / 30.), tEnd(1), substeps(10), bodies(nullptr), body_count(0),
      constraints(nullptr), constraint_count(0), constraint_layers(nullptr),
      layer_count(0), constraint_layer_sizes(nullptr), body_layers(nullptr),
      body_layer_sizes(nullptr), gravity(0.0, 0.0, -980.0), iters(1),
      ground_E(Eigen::Matrix4f::Zero()), ground_size(10),
      axis(Eigen::Matrix<float, 6, 1>::Zero()), steps(0) {}

Model::Model(const Model &other)
    : h(other.h), tEnd(other.tEnd), substeps(other.substeps), bodies(nullptr),
      body_count(other.body_count), constraints(other.constraints),
      constraint_count(other.constraint_count), constraint_layers(nullptr),
      layer_count(other.layer_count), constraint_layer_sizes(nullptr),
      body_layers(nullptr), body_layer_sizes(nullptr), gravity(other.gravity),
      iters(other.iters), ground_E(other.ground_E),
      ground_size(other.ground_size), axis(other.axis), steps(other.steps) {
#ifdef __CUDA_ARCH__
  // we are on the device; don't copy the bodies
#else
  hipPointerAttribute_t attributes;
  CUDA_CHECK(hipPointerGetAttributes(&attributes, other.bodies));
  if (attributes.type == hipMemoryTypeDevice) {
    bodies = alloc_device<Body>(other.body_count);
    memcpy_device(bodies, other.bodies, other.body_count);
  } else {
    size_t size = other.body_count * sizeof(Body);
    bodies = new Body[other.body_count];
    memcpy(bodies, other.bodies, size);
  }
#endif
}

void Model::init() {
  // initialize bodies
  // if (this->bodies == nullptr) {
  //   throw std::runtime_error("Bodies not initialized");
  // }
  for (size_t i = 0; i < this->body_count; i++) {
    this->bodies[i].init();
  }
  // create constraints
  // if (this->constraints == nullptr && this->constraint_count) {
  //   throw std::runtime_error("Constraints not initialized");
  // }
  for (size_t i = 0; i < this->constraint_count; i++) {
    this->constraints[i].init();
  }
  // calculate parameters
  this->steps = ceil(this->tEnd / this->h);
  // this->hs = this->h / this->substeps;
  // this->k = 0;
  // this->ks
}

void Model::move_to_device() {
  bodies = move_array_to_device(bodies, body_count);
  constraints = move_array_to_device(constraints, constraint_count);
  // TODO: layers
}

__device__ Model Model::clone_with_buffers(const Model &other, size_t scene_id,
                                           Body *body_buffer) {
  Model new_model(other);

  new_model.bodies = &body_buffer[other.body_count * scene_id];
  memcpy_device(new_model.bodies, other.bodies, other.body_count);
  return new_model;
}

void Model::simulate(Collider *collider) {
  float time = 0;
  float hs = this->h / static_cast<float>(this->substeps);
  // std::cout << this->steps << ", " << this->substeps << std::endl;
  for (unsigned int step = 0; step < this->steps; step++) {
    this->clearBodyShockPropInfo();
    collider->run(this);
    this->constructConstraintGraph();
    for (unsigned int substep = 0; substep < this->substeps; substep++) {
      this->stepBDF1(step, substep, hs);
      this->solveConSP(hs);
      this->solveConGS(hs);
      time += hs;
    }
    this->computeEnergies();
    // draw ?
    this->write_state(step);
  }
}

/** Private Functions **/

void Model::stepBDF1(unsigned int step, unsigned int substep, float hs) {
  for (size_t body_i = 0; body_i < this->body_count; body_i++) {
    this->bodies[body_i].stepBDF1(step, substep, hs, this->gravity);
  }
}
void Model::clearBodyShockPropInfo() {
  // TODO
  // clears the shock propagation info from each body; this may not be necessary
  // depending on implementation
  for (size_t body_i = 0; body_i < this->body_count; body_i++) {
    this->bodies[body_i].clearShock();
  }
}
void Model::constructConstraintGraph() {
  // TODO
  // Constructs a graph of constraints, working from the ground layer up
  // needs a list of constraints and bodies
  // constraints needs:
  //  - list of bodies
  // body needs:
  //  - layer
  //  - shock parent constraint
  //  - constraint
  //
  //  collect static constraints and collision constraints
  //  for each constraint:
  //    if it is ground, initialize the body affected to layer 1 and add this
  //    constraint to the list affecting that body if the constraint has 2
  //    bodies (not 1), then tell both bodies that this constraint affects them
  //
  //  working up one layer at a time:
  //    for every body affected in the previous layer:
  //      for each constraint affecting the body:
  //        make sure the body on a higher layer is second
  //        assign the second body to this layer, and add this constraint to the
  //        parent constraints add the second body and this constraint to this
  //        layer

  // in theory, we can do this by keeping a constraint and body layer list, and
  // a last layer constraint list. the shock parent list is more difficult, but
  // does not seem to be used for anything other than setting shockProp to true;
  // so we can do this here
}
void Model::solveConSP(float hs) {
  for (size_t constraint_i = 0; constraint_i < this->constraint_count;
       constraint_i++) {
    this->constraints[constraint_i].clear();
  }

  for (size_t i = 0; i < this->layer_count; i++) {
    for (int iter = 0; iter < this->iters; iter++) {
      for (size_t j = 0; j < this->constraint_layer_sizes[i]; j++) {
        this->constraints[this->constraint_layers[i * MAX_LAYER_SIZE + j]]
            .solve(hs, true);
      }
    }
  }

  for (long i = this->layer_count - 1; i >= 0; i--) {
    for (size_t j = 0; j < this->body_layer_sizes[i]; j++) {
      this->bodies[this->body_layers[i * MAX_LAYER_SIZE + j]]
          .applyJacobiShock();
    }
    for (int iter = 0; iter < this->iters; iter++) {
      for (size_t j = 0; j < this->constraint_layer_sizes[i]; j++) {
        this->constraints[this->constraint_layers[i * MAX_LAYER_SIZE + j]]
            .solve(hs, false);
      }
    }
  }
}
void Model::solveConGS(float hs) {
  for (int iter = 0; iter < this->iters; iter++) {
    for (size_t constraint_i = 0; constraint_i < this->constraint_count;
         constraint_i++) {
      this->constraints[constraint_i].solve(hs, false);
    }
  }
}
void Model::computeEnergies() { /*TODO*/
}

void Model::write_state(unsigned int step) {
#ifdef WRITE
  printf("Step %d\n", step);
  for (size_t i = 0; i < body_count; i++) {
    printf("%d ", i);
    bodies[i].write_state();
    printf("\n");
  }
#endif
}

} // namespace apbd
