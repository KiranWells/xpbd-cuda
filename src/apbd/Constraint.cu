#include "hip/hip_runtime.h"
#include "util.h"
#include <apbd/Collider.h>
#include <apbd/Constraint.h>
#include <se3/lib.h>

using Eigen::Vector2f, Eigen::Vector3f, Eigen::Vector4f, Eigen::Quaternionf;

namespace apbd {

Constraint::Constraint(ConstraintRigid rigid)
    : type(CONSTRAINT_COLLISION_RIGID), data{.rigid = {rigid}} {}
Constraint::Constraint(ConstraintGround ground)
    : type(CONSTRAINT_COLLISION_GROUND), data{.ground = {ground}} {}
Constraint::Constraint(ConstraintJointRevolve revolve)
    : type(CONSTRAINT_JOINT_REVOLVE), data{.joint_revolve = {revolve}} {}

ConstraintGround::ConstraintGround(BodyRigid *body, Eigen::Matrix4f Eg, float d,
                                   Eigen::Vector3f xl, Eigen::Vector3f xw,
                                   Eigen::Vector3f nw, Eigen::Vector3f vw)
    : C(Vector3f::Zero()), lambda(Vector3f::Zero()), nw(nw),
      lambdaSF(Vector3f::Zero()), d(0), dlambdaNor(0), shockProp(false),
      body(body), Eg(Eg), xl(xl), xw(xw), vw(vw) {}

ConstraintRigid::ConstraintRigid(BodyRigid *body1, BodyRigid *body2, float d,
                                 Eigen::Vector3f nw, Eigen::Vector3f x1,
                                 Eigen::Vector3f x2)
    : C(Vector3f::Zero()), lambda(Vector3f::Zero()), nw(nw),
      lambdaSF(Vector3f::Zero()), d(0), dlambdaNor(0), shockProp(false),
      body1(body1), body2(body2), x1(x1), x2(x2) {}

Constraint &Constraint::operator=(const Constraint &other) {
  this->type = other.type;
  switch (type) {
  case CONSTRAINT_COLLISION_GROUND: {
    this->data.ground = other.data.ground;
    break;
  }
  case CONSTRAINT_COLLISION_RIGID: {
    this->data.rigid = other.data.rigid;
    break;
  }
  case CONSTRAINT_JOINT_REVOLVE: {
    this->data.joint_revolve = other.data.joint_revolve;
    break;
  }

  default:
    break;
  }
  return *this;
}

void Constraint::init() {
  // TODO: handle joint_revolve
}

void Constraint::clear() {
  switch (type) {
  case CONSTRAINT_COLLISION_GROUND: {
    ConstraintGround *c = &data.ground;
    c->C = Eigen::Vector3f::Zero();
    c->lambda = Eigen::Vector3f::Zero();
    break;
  }
  case CONSTRAINT_COLLISION_RIGID: {
    ConstraintRigid *c = &data.rigid;
    c->C = Eigen::Vector3f::Zero();
    c->lambda = Eigen::Vector3f::Zero();
    break;
  }
  case CONSTRAINT_JOINT_REVOLVE: {
    ConstraintJointRevolve *c = &data.joint_revolve;
    c->C = Eigen::Vector3f::Zero();
    c->lambda = Eigen::Vector3f::Zero();
    break;
  }

  default:
    break;
  }
}

void Constraint::solve(float hs, bool doShockProp) {
  switch (type) {
  case CONSTRAINT_COLLISION_GROUND: {
    ConstraintGround *c = &data.ground;
    c->solveNorPos(hs);
    c->applyJacobi();
    break;
  }
  case CONSTRAINT_COLLISION_RIGID: {
    ConstraintRigid *c = &data.rigid;
    c->solveNorPos(hs);
    c->applyJacobi();
    break;
  }
  case CONSTRAINT_JOINT_REVOLVE: {
    ConstraintJointRevolve *c = &data.joint_revolve;
    c->solve();
    break;
  }

  default:
    break;
  }
}

void ConstraintGround::solveNorPos(float hs) {
  Vector3f v = hs * body->computePointVel(xl, hs);
  float vNorm = v.norm();
  Vector3f vNormalized = v / vNorm;
  Vector3f tx = Eg.block<3, 1>(0, 0);
  Vector3f ty = Eg.block<3, 1>(0, 1);
  Eigen::Matrix3f frame_tmp;
  frame_tmp << nw, tx, ty;
  Vector3f vNormalizedContactFrame = frame_tmp.transpose() * vNormalized;

  float dlambda = solvePosDir1(vNorm, vNormalized);
  C = vNorm * vNormalizedContactFrame;

  float dlambdaNor = dlambda * vNormalizedContactFrame(0);
  float lambdaNor = lambda(0) + dlambdaNor;
  if (lambdaNor < 0) {
    dlambdaNor = -lambda(0);
  }
  lambda(0) += dlambdaNor;
  float mu = body->mu;
  Vector2f dlambdaTan = Vector2f::Zero();
  if (mu > 0) {
    float dlambdaTx = dlambda * vNormalizedContactFrame(1);
    float dlambdaTy = dlambda * vNormalizedContactFrame(2);
    float lambdaNorLenMu = mu * lambda(0);
    Vector2f lambdaTan = Vector2f(lambda(1) + dlambdaTx, lambda(2) + dlambdaTy);
    float lambdaTanLen = lambdaTan.norm();
    dlambdaTan = Vector2f(dlambdaTx, dlambdaTy);
    if (lambdaTanLen > lambdaNorLenMu) {
      dlambdaTan = (lambdaTan / lambdaTanLen * lambdaNorLenMu -
                    Vector2f(lambda(1), lambda(2)));
    }
    lambda(1) += dlambdaTan(0);
    lambda(2) += dlambdaTan(1);
  }

  Vector3f frictionalContactLambda =
      Vector3f(dlambdaNor, dlambdaTan(0), dlambdaTan(1));
  dlambda = frictionalContactLambda.norm();
  if (dlambda > 0) {
    // frictionalContactNormal = [this->nw, tx, ty] * frictionalContactLambda ./
    // dlambda;
    Eigen::Matrix3f tmp;
    tmp << nw, tx, ty;
    Vector3f frictionalContactNormal = tmp * frictionalContactLambda / dlambda;
    vec7 dq = computeDx(dlambda, frictionalContactNormal);
    body->dxJacobi.block<4, 1>(0, 0) += dq.block<4, 1>(0, 0);
    body->dxJacobi.block<3, 1>(4, 0) += dq.block<3, 1>(4, 0);
  }
}

float ConstraintGround::solvePosDir1(float c, Eigen::Vector3f nw) {
  // Use the provided normal rather than normalizing
  auto m1 = this->body->Mp;
  auto I1 = this->body->Mr;
  Quaternionf q1 = Quaternionf(this->body->x.block<4, 1>(0, 0));
  Vector3f nl1 = se3::invert_q(q1) * nw;
  Vector3f rl1 = this->xl;
  Vector3f rnl1 = rl1.cross(nl1);
  float w1 = (1 / m1) + rnl1.transpose() * Vector3f(rnl1.array() / I1.array());
  float numerator = -c;
  float denominator = w1;
  return numerator / denominator;
}

vec7 ConstraintGround::computeDx(float dlambda, Eigen::Vector3f nw) {

  float m1 = body->Mp;
  Vector3f I1 = body->Mr;
  // Position update
  Vector3f dpw = dlambda * nw;
  Vector3f dp = dpw / m1;
  // Quaternion update
  Quaternionf q1 = Quaternionf(Eigen::Vector4f(body->x1_0.block<4, 1>(0, 0)));
  auto dpl1 = se3::qRotInv(q1.coeffs(), dpw);
  // auto dpl1 = (se3::invert_q(q1) * dpw);
  Vector4f q2vec;
  q2vec << se3::qRot(q1.coeffs(), (xl.cross(dpl1).array() / I1.array())), 0;
  // q2vec << (q1 * (xl.cross(dpl1).array() / I1.array())), 0;
  // qtmp1 = [I1.\se3.cross(rl1,dpl1); 0];
  // dq = se3.qMul(sin(0.5*qtmp1),q1);
  Quaternionf q2(q2vec);
  Vector4f dq = 0.5 * se3::qMul(q2.coeffs(), q1.coeffs());
  // Vector4f dq = 0.5 * (q2 * q1).coeffs();
  vec7 out;
  out << dq, dp;
  return out;
}

void ConstraintGround::applyJacobi() { body->applyJacobi(); }
void ConstraintRigid::applyJacobi() {
  body1->applyJacobi();
  body2->applyJacobi();
}

void ConstraintRigid::solveNorPos(float hs) {

  Vector3f v1w = this->body1->computePointVel(this->x1, hs);
  Vector3f v2w = this->body2->computePointVel(this->x2, hs);
  Vector3f v = hs * (v1w - v2w);
  float vNorm = v.norm();
  Vector3f vNormalized = v / vNorm;
  Vector3f tx, ty;
  Collider::generateTangents(this->nw, &tx, &ty);
  // vNormalizedContactFrame = [-this->nw'; tx' ; ty'] * vNormalized;
  Eigen::Matrix3f tmp;
  tmp << -this->nw, tx, ty;
  Vector3f vNormalizedContactFrame = tmp.transpose() * vNormalized;

  float dlambda = this->solvePosDir2(vNorm, vNormalized);
  this->C = vNorm * vNormalizedContactFrame;

  float dlambdaNor = dlambda * vNormalizedContactFrame(0);
  float lambdaNor = this->lambda(0) + dlambdaNor;
  if (lambdaNor < 0) {
    dlambdaNor = -this->lambda(0);
  }
  this->lambda(0) = this->lambda(0) + dlambdaNor;
  float mu1 = this->body1->mu;
  float mu2 = this->body2->mu;
  float mu = 0.5 * (mu1 + mu2);
  Vector2f dlambdaTan{0, 0};
  if (mu > 0) {
    float dlambdaTx = dlambda * vNormalizedContactFrame(1);
    float dlambdaTy = dlambda * vNormalizedContactFrame(2);
    float lambdaNorLenMu = mu * this->lambda(0);
    Vector2f lambdaTan{this->lambda(1) + dlambdaTx,
                       this->lambda(2) + dlambdaTy};
    float lambdaTanLen = lambdaTan.norm();
    dlambdaTan = Vector2f(dlambdaTx, dlambdaTy);
    if (lambdaTanLen > lambdaNorLenMu) {
      dlambdaTan = lambdaTan / lambdaTanLen * lambdaNorLenMu -
                   Vector2f(this->lambda(1), this->lambda(2));
    }
    this->lambda(1) = this->lambda(1) + dlambdaTan(0);
    this->lambda(2) = this->lambda(2) + dlambdaTan(1);
  }

  Vector3f frictionalContactLambda;
  frictionalContactLambda << dlambdaNor, dlambdaTan;
  dlambda = frictionalContactLambda.norm();
  if (dlambda > 0) {
    Eigen::Matrix3f tmp;
    tmp << -this->nw, tx, ty;
    Vector3f frictionalContactNormal = tmp * frictionalContactLambda / dlambda;
    Vector4f dq1, dq2;
    Vector3f dp1, dp2;
    this->computeDx(dlambda, frictionalContactNormal, &dq1, &dp1, &dq2, &dp2);
    if (this->shockProp) {
      this->body1->dxJacobiShock.block<4, 1>(0, 0) += dq1;
      this->body1->dxJacobiShock.block<3, 1>(4, 0) += dp1;
    } else {
      this->body1->dxJacobi.block<4, 1>(0, 0) += dq1;
      this->body1->dxJacobi.block<3, 1>(4, 0) += dp1;
    }
    this->body2->dxJacobi.block<4, 1>(0, 0) += dq2;
    this->body2->dxJacobi.block<3, 1>(4, 0) += dp2;
  }
}
float ConstraintRigid::solvePosDir2(float c, Eigen::Vector3f nw) {
  // Use the provided normal rather than normalizing
  auto m1 = this->body1->Mp;
  auto m2 = this->body2->Mp;
  auto I1 = this->body1->Mr;
  auto I2 = this->body2->Mr;
  Quaternionf q1 = Quaternionf(this->body1->x.block<4, 1>(0, 0));
  Quaternionf q2 = Quaternionf(this->body2->x.block<4, 1>(0, 0));
  Vector3f nl1 = se3::invert_q(q1) * nw;
  Vector3f nl2 = se3::invert_q(q2) * nw;
  Vector3f rl1 = this->x1;
  Vector3f rl2 = this->x2;
  Vector3f rnl1 = rl1.cross(nl1);
  Vector3f rnl2 = rl2.cross(nl2);
  float w1 = (1 / m1) + rnl1.transpose() * Vector3f(rnl1.array() / I1.array());
  float w2 = (1 / m2) + rnl2.transpose() * Vector3f(rnl2.array() / I2.array());
  float numerator = -c;
  float denominator = w1 + w2;
  return numerator / denominator;
}

void ConstraintRigid::computeDx(float dlambda, Eigen::Vector3f nw,
                                Vector4f *dq1, Vector3f *dp1, Vector4f *dq2,
                                Vector3f *dp2) {
  auto m1 = this->body1->Mp;
  auto m2 = this->body2->Mp;
  auto I1 = this->body1->Mr;
  auto I2 = this->body2->Mr;
  // Position update
  Vector3f dpw = dlambda * nw;
  *dp1 = dpw / m1;
  *dp2 = -dpw / m2;
  // Quaternion update
  Quaternionf q1 = Quaternionf(this->body1->x1_0.block<4, 1>(0, 0));
  Quaternionf q2 = Quaternionf(this->body2->x1_0.block<4, 1>(0, 0));
  Vector3f dpl1 = se3::qRotInv(q1.coeffs(), dpw);
  Vector3f dpl2 = se3::qRotInv(q2.coeffs(), dpw);
  // Vector3f dpl1 = se3::invert_q(q1) * dpw;
  // Vector3f dpl2 = se3::invert_q(q2) * dpw;

  // qtmp1 = [se3.qRot(q1,I1.\se3.cross(this.x1,dpl1)); 0];
  Vector3f tmp =
      se3::qRot(q1.coeffs(), (this->x1.cross(dpl1).array() / I1.array()));
  // Vector3f tmp = q1 * (this->x1.cross(dpl1).array() / I1.array());
  Quaternionf qtmp1(0, tmp.x(), tmp.y(), tmp.z());

  // qtmp2 = [se3.qRot(q2,I2.\se3.cross(this.x2,dpl2)); 0];
  Vector3f tmp1 =
      se3::qRot(q2.coeffs(), (this->x2.cross(dpl2).array() / I2.array()));
  // Vector3f tmp1 = q2 * (this->x2.cross(dpl2).array() / I2.array());
  Quaternionf qtmp2(0, tmp1.x(), tmp1.y(), tmp1.z());

  // dq1 = se3.qMul(sin(0.5*qtmp1),q1);
  // dq2 = se3.qMul(sin(-0.5*qtmp2),q2);
  // Vector4f dq1 = (Quaternionf(Vector4f((0.5 * qtmp1.coeffs()).array().sin()))
  // * q1).coeffs(); Vector4f dq2 = (Quaternionf(Vector4f((-0.5 *
  // qtmp2.coeffs()).array().sin())) * q2).coeffs();
  *dq1 = 0.5 * se3::qMul(qtmp1.coeffs(), q1.coeffs());
  *dq2 = -0.5 * se3::qMul(qtmp2.coeffs(), q2.coeffs());
  // Vector4f dq1 = 0.5 * (qtmp1 * q1).coeffs();
  // Vector4f dq2 = -0.5 * (qtmp2 * q2).coeffs();
}

void ConstraintJointRevolve::solve() {
  // TODO
}
} // namespace apbd
