#include "hip/hip_runtime.h"
#include "apbd/Collider.h"
#include "util.h"

namespace apbd {

Collider::Collider(Model *model)
    : bp_cap_1(model->body_count),
      bp_cap_2(model->body_count * model->body_count), bp_count_1(0),
      bp_count_2(0), bpList1(nullptr), bpList2(nullptr), collision_count(0),
      collisions(nullptr) {
  bpList1 = alloc_device<Body *>(bp_cap_1);
  bpList2 = alloc_device<Body *>(bp_cap_2);
  collisions = alloc_device<Constraint>(MAX_COLLISIONS);
}

Collider::Collider(Model *model, size_t scene_id, Body **body_ptr_buffer,
                   Constraint *constraint_buffer)
    : bp_cap_1(model->body_count),
      bp_cap_2(model->body_count * model->body_count), bp_count_1(0),
      bp_count_2(0), bpList1(&body_ptr_buffer[model->body_count * scene_id *
                                              (model->body_count + 1)]),
      bpList2(&body_ptr_buffer[model->body_count * scene_id *
                                   (model->body_count + 1) +
                               model->body_count]),
      collision_count(0),
      collisions(&constraint_buffer[MAX_COLLISIONS * scene_id]) {}
void Collider::allocate_buffers(Model &model, int sim_count,
                                Body **&body_ptr_buffer,
                                Constraint *&constraint_buffer) {
  body_ptr_buffer = alloc_device<Body *>(model.body_count *
                                         (model.body_count + 1) * sim_count);
  constraint_buffer = alloc_device<Constraint>(MAX_COLLISIONS * sim_count);
}

void Collider::run(Model *model) {
  bp_count_1 = 0;
  bp_count_2 = 0;
  collision_count = 0;
  this->broadphase(model);
  this->narrowphase(model);
}

void Collider::broadphase(Model *model) {
  Body *bodies = model->bodies;

  for (size_t i = 0; i < model->body_count && this->bp_count_1 < this->bp_cap_1;
       i++) {
    Body *body = &bodies[i];
    if (body->collide()) {
      if (body->broadphaseGround(model->ground_E)) {
        this->bpList1[this->bp_count_1++] = body;
      }
    }
  }
  for (size_t i = 0; i < model->body_count && this->bp_count_2 < this->bp_cap_2;
       i++) {
    Body *body = &bodies[i];
    if (body->collide()) {
      for (size_t j = i + 1;
           j < model->body_count && this->bp_count_2 < this->bp_cap_2; j++) {
        if (bodies[j].collide()) {
          if (body->broadphaseRigid(&bodies[j])) {
            this->bpList2[this->bp_count_2++] = body;
            this->bpList2[this->bp_count_2++] = &bodies[j];
          }
        }
      }
    }
  }
}

void Collider::narrowphase(Model *model) {
  auto Eg = model->ground_E;

  for (size_t i = 0; i < this->bp_count_1; i++) {
    auto body = this->bpList1[i];
    auto cpair = body->narrowphaseGround(Eg);
    auto cdata = cpair.first;
    auto c_count = cpair.second;
    for (size_t k = 0; k < c_count && this->collision_count < MAX_COLLISIONS;
         k++) {
      auto &c = cdata[k];
      switch (body->type) {
      case BODY_RIGID: {
        this->collisions[this->collision_count++] = Constraint(ConstraintGround(
            &body->data.rigid, Eg, c.d, c.xl, c.xw, c.nw, c.vw));
        break;
      }
      default:
        break;
      }
    }
  }

  for (size_t i = 0; i < this->bp_count_2; i += 2) {
    auto body1 = this->bpList2[i];
    auto body2 = this->bpList2[i + 1];
    auto cpair = body1->narrowphaseRigid(body2);
    auto cdata = cpair.first;
    auto c_count = cpair.second;
    for (size_t k = 0; k < c_count; k++) {
      auto &c = cdata[k];
      switch (body1->type) {
      case BODY_RIGID: {
        // we require the other body to be rigid
        if (body2->type == BODY_RIGID) {
          this->collisions[this->collision_count++] =
              Constraint(ConstraintRigid(&body1->data.rigid, &body2->data.rigid,
                                         c.d, c.nw, c.x1, c.x2));
        }
        break;
      }
      default:
        break;
      }
    }
  }
}

std::pair<Eigen::Vector3f, Eigen::Vector3f>
Collider::generateTangents(Eigen::Vector3f nor) {
  Eigen::Vector3f tmp;
  if (abs(nor(2)) < 1e-6) {
    tmp << 0, 0, 1;
  } else {
    tmp << 1, 0, 0;
  }
  Eigen::Vector3f tany = nor.cross(tmp);
  tany = tany / tany.norm();
  Eigen::Vector3f tanx = tany.cross(nor);
  return std::pair(tanx, tany);
}

} // namespace apbd
