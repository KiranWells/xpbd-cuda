#include "hip/hip_runtime.h"
#include "apbd/BodyReference.h"
#include "model_samples.h"
#include <exception>
#include <getopt.h>
#include <iostream>
#include <string>
#include <thread>
#include <vector>

using std::cout, std::endl, std::string, std::runtime_error;
typedef std::chrono::high_resolution_clock Clock;

__global__ void kernel(apbd::Model model, apbd::Body *body_buffer,
                       apbd::Body **body_ptr_buffer,
                       apbd::Constraint *constraint_buffer, int sims) {
  // get this scene ID
  size_t scene_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (scene_id >= sims)
    return;
  model.copy_data_to_store();
  auto r = apbd::BodyReference(0, apbd::BODY_RIGID);
  auto b = r.get_rigid();

  DEBUG_VEC(b.xdotInit(), 7);

  // // make a copy of the model
  // apbd::Model thread_model =
  //     apbd::Model::clone_with_buffers(model, scene_id, body_buffer);
  // // create a thread-local collider
  // auto collider = apbd::Collider(&thread_model, scene_id, body_ptr_buffer,
  //                                constraint_buffer);
  // // simulate
  // thread_model.simulate(&collider);
}

void run_kernel(apbd::Model model, int sims) {
  cout << "thread blocks: " << (sims + BLOCK_SIZE - 1) / BLOCK_SIZE << endl;

  // const size_t shared_size = c.constraints.size() * sizeof(Constraint);
  const size_t shared_size = 0;
  // std::cout << "kernel shared_size: " << shared_size << " = " <<
  // c.constraints.size() << " * " << sizeof(Constraint) << std::endl;

  size_t body_buffer_size = sims * model.body_count;
  apbd::Body *body_buffer = alloc_device<apbd::Body>(body_buffer_size);
  apbd::Body **body_ptr_buffer = nullptr;
  apbd::Constraint *constraint_buffer = nullptr;
  apbd::Collider::allocate_buffers(model, sims, body_ptr_buffer,
                                   constraint_buffer);

  model.move_to_device();

  std::cout << "kernel start" << std::endl;
  auto t1 = Clock::now();

  kernel<<<(sims + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, shared_size>>>(
      model, body_buffer, body_ptr_buffer, constraint_buffer, sims);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  auto t2 = Clock::now();
  std::cout << "Kernel took: " << (t2 - t1).count() << '\t';
}

int main(int argc, char *argv[]) {
  // auto state = parse_arguments(argc, argv);
  auto model = createModelSample(atoi(argv[1]));
  model.create_store();

  // auto t1 = Clock::now();
#ifdef USE_CUDA
  cout << "Running with CUDA" << endl;
  run_kernel(model, 1);
#else
  // cout << "Running on CPU" << endl;
  // cpu_run_group(model, state.scene_count);
#endif
  // auto t2 = Clock::now();
  // cout << "Simulation took: " << (t2 - t1).count() << '\n';
}
